#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <time.h>
#include <signal.h>
#include <unistd.h>
#include <stdint.h> 





__device__ unsigned char gmul_table[256][256];

#define DEBUG_MODE  0

#define TIMER 0



__constant__ unsigned char const_sBox[256] = {
    0x63, 0x7c, 0x77, 0x7b, 0xf2, 0x6b, 0x6f, 0xc5, 0x30, 0x01, 0x67, 0x2b, 0xfe, 0xd7, 0xab, 0x76,
    0xca, 0x82, 0xc9, 0x7d, 0xfa, 0x59, 0x47, 0xf0, 0xad, 0xd4, 0xa2, 0xaf, 0x9c, 0xa4, 0x72, 0xc0,
    0xb7, 0xfd, 0x93, 0x26, 0x36, 0x3f, 0xf7, 0xcc, 0x34, 0xa5, 0xe5, 0xf1, 0x71, 0xd8, 0x31, 0x15,
    0x04, 0xc7, 0x23, 0xc3, 0x18, 0x96, 0x05, 0x9a, 0x07, 0x12, 0x80, 0xe2, 0xeb, 0x27, 0xb2, 0x75,
    0x09, 0x83, 0x2c, 0x1a, 0x1b, 0x6e, 0x5a, 0xa0, 0x52, 0x3b, 0xd6, 0xb3, 0x29, 0xe3, 0x2f, 0x84,
    0x53, 0xd1, 0x00, 0xed, 0x20, 0xfc, 0xb1, 0x5b, 0x6a, 0xcb, 0xbe, 0x39, 0x4a, 0x4c, 0x58, 0xcf,
    0xd0, 0xef, 0xaa, 0xfb, 0x43, 0x4d, 0x33, 0x85, 0x45, 0xf9, 0x02, 0x7f, 0x50, 0x3c, 0x9f, 0xa8,
    0x51, 0xa3, 0x40, 0x8f, 0x92, 0x9d, 0x38, 0xf5, 0xbc, 0xb6, 0xda, 0x21, 0x10, 0xff, 0xf3, 0xd2,
    0xcd, 0x0c, 0x13, 0xec, 0x5f, 0x97, 0x44, 0x17, 0xc4, 0xa7, 0x7e, 0x3d, 0x64, 0x5d, 0x19, 0x73,
    0x60, 0x81, 0x4f, 0xdc, 0x22, 0x2a, 0x90, 0x88, 0x46, 0xee, 0xb8, 0x14, 0xde, 0x5e, 0x0b, 0xdb,
    0xe0, 0x32, 0x3a, 0x0a, 0x49, 0x06, 0x24, 0x5c, 0xc2, 0xd3, 0xac, 0x62, 0x91, 0x95, 0xe4, 0x79,
    0xe7, 0xc8, 0x37, 0x6d, 0x8d, 0xd5, 0x4e, 0xa9, 0x6c, 0x56, 0xf4, 0xea, 0x65, 0x7a, 0xae, 0x08,
    0xba, 0x78, 0x25, 0x2e, 0x1c, 0xa6, 0xb4, 0xc6, 0xe8, 0xdd, 0x74, 0x1f, 0x4b, 0xbd, 0x8b, 0x8a,
    0x70, 0x3e, 0xb5, 0x66, 0x48, 0x03, 0xf6, 0x0e, 0x61, 0x35, 0x57, 0xb9, 0x86, 0xc1, 0x1d, 0x9e,
    0xe1, 0xf8, 0x98, 0x11, 0x69, 0xd9, 0x8e, 0x94, 0x9b, 0x1e, 0x87, 0xe9, 0xce, 0x55, 0x28, 0xdf,
    0x8c, 0xa1, 0x89, 0x0d, 0xbf, 0xe6, 0x42, 0x68, 0x41, 0x99, 0x2d, 0x0f, 0xb0, 0x54, 0xbb, 0x16
};



__constant__ unsigned char const_invSBox[256] = {
    0x52, 0x09, 0x6a, 0xd5, 0x30, 0x36, 0xa5, 0x38, 0xbf, 0x40, 0xa3, 0x9e, 0x81, 0xf3, 0xd7, 0xfb,
    0x7c, 0xe3, 0x39, 0x82, 0x9b, 0x2f, 0xff, 0x87, 0x34, 0x8e, 0x43, 0x44, 0xc4, 0xde, 0xe9, 0xcb,
    0x54, 0x7b, 0x94, 0x32, 0xa6, 0xc2, 0x23, 0x3d, 0xee, 0x4c, 0x95, 0x0b, 0x42, 0xfa, 0xc3, 0x4e,
    0x08, 0x2e, 0xa1, 0x66, 0x28, 0xd9, 0x24, 0xb2, 0x76, 0x5b, 0xa2, 0x49, 0x6d, 0x8b, 0xd1, 0x25,
    0x72, 0xf8, 0xf6, 0x64, 0x86, 0x68, 0x98, 0x16, 0xd4, 0xa4, 0x5c, 0xcc, 0x5d, 0x65, 0xb6, 0x92,
    0x6c, 0x70, 0x48, 0x50, 0xfd, 0xed, 0xb9, 0xda, 0x5e, 0x15, 0x46, 0x57, 0xa7, 0x8d, 0x9d, 0x84,
    0x90, 0xd8, 0xab, 0x00, 0x8c, 0xbc, 0xd3, 0x0a, 0xf7, 0xe4, 0x58, 0x05, 0xb8, 0xb3, 0x45, 0x06,
    0xd0, 0x2c, 0x1e, 0x8f, 0xca, 0x3f, 0x0f, 0x02, 0xc1, 0xaf, 0xbd, 0x03, 0x01, 0x13, 0x8a, 0x6b,
    0x3a, 0x91, 0x11, 0x41, 0x4f, 0x67, 0xdc, 0xea, 0x97, 0xf2, 0xcf, 0xce, 0xf0, 0xb4, 0xe6, 0x73,
    0x96, 0xac, 0x74, 0x22, 0xe7, 0xad, 0x35, 0x85, 0xe2, 0xf9, 0x37, 0xe8, 0x1c, 0x75, 0xdf, 0x6e,
    0x47, 0xf1, 0x1a, 0x71, 0x1d, 0x29, 0xc5, 0x89, 0x6f, 0xb7, 0x62, 0x0e, 0xaa, 0x18, 0xbe, 0x1b,
    0xfc, 0x56, 0x3e, 0x4b, 0xc6, 0xd2, 0x79, 0x20, 0x9a, 0xdb, 0xc0, 0xfe, 0x78, 0xcd, 0x5a, 0xf4,
    0x1f, 0xdd, 0xa8, 0x33, 0x88, 0x07, 0xc7, 0x31, 0xb1, 0x12, 0x10, 0x59, 0x27, 0x80, 0xec, 0x5f,
    0x60, 0x51, 0x7f, 0xa9, 0x19, 0xb5, 0x4a, 0x0d, 0x2d, 0xe5, 0x7a, 0x9f, 0x93, 0xc9, 0x9c, 0xef,
    0xa0, 0xe0, 0x3b, 0x4d, 0xae, 0x2a, 0xf5, 0xb0, 0xc8, 0xeb, 0xbb, 0x3c, 0x83, 0x53, 0x99, 0x61,
    0x17, 0x2b, 0x04, 0x7e, 0xba, 0x77, 0xd6, 0x26, 0xe1, 0x69, 0x14, 0x63, 0x55, 0x21, 0x0c, 0x7d,
};



__constant__ unsigned char const_rcon[10] = {
    0x01, 0x02, 0x04, 0x08, 0x10, 0x20, 0x40, 0x80, 0x1b, 0x36
};



__device__ void KeyExpansion(unsigned char* key, unsigned char* roundKeys) {
    int i, j;
    unsigned char temp[4], k;

    for (i = 0; i < 16; i++) {
        roundKeys[i] = key[i];
    }

    for (; i < 176; i += 4) {
        for (j = 0; j < 4; j++) {
            temp[j] = roundKeys[i - 4 + j];
        }

        if (i % 16 == 0) {
            k = temp[0];
            temp[0] = temp[1];
            temp[1] = temp[2];
            temp[2] = temp[3];
            temp[3] = k;

            for (j = 0; j < 4; j++) {
                temp[j] = const_sBox[temp[j]];
            }

            temp[0] = temp[0] ^ const_rcon[i / 16 - 1];
        }

        for (j = 0; j < 4; j++) {
            roundKeys[i + j] = roundKeys[i - 16 + j] ^ temp[j];
        }
    }
}



__device__ void SubBytes(unsigned char* state) {
    for (int i = 0; i < 16; i++) {
        state[i] = const_sBox[state[i]];
    }
}


__device__ void InvSubBytes(unsigned char* state) {
    for (int i = 0; i < 16; i++) {
        state[i] = const_invSBox[state[i]];
    }
}



__device__ void ShiftRows(unsigned char* state) {
    unsigned char temp;




    temp = state[1];
    state[1] = state[5];
    state[5] = state[9];
    state[9] = state[13];
    state[13] = temp;


    temp = state[2];
    state[2] = state[10];
    state[10] = temp;
    temp = state[6];
    state[6] = state[14];
    state[14] = temp;


    temp = state[3];
    state[3] = state[15];
    state[15] = state[11];
    state[11] = state[7];
    state[7] = temp;
}




__device__ void InvShiftRows(unsigned char* state) {
    unsigned char temp;

    temp = state[13];
    state[13] = state[9];
    state[9] = state[5];
    state[5] = state[1];
    state[1] = temp;


    temp = state[2];
    state[2] = state[10];
    state[10] = temp;
    temp = state[6];
    state[6] = state[14];
    state[14] = temp;


    temp = state[3];
    state[3] = state[7];
    state[7] = state[11];
    state[11] = state[15];
    state[15] = temp;
}


__device__ unsigned char gmul(unsigned char a, unsigned char b) {

    unsigned char p = 0;
    unsigned char high_bit_mask = 0x80;
    unsigned char high_bit;
    unsigned char modulo = 0x1B; /* x^8 + x^4 + x^3 + x + 1 */

    for (int i = 0; i < 8; i++) {
        if (b & 1) {
            p ^= a;
        }
        high_bit = a & high_bit_mask;
        a <<= 1;
        if (high_bit) {
            a ^= modulo;
        }
        b >>= 1;
    }
    return p;
}


__global__ void precompute_gmul_table_aes128() {
    for (int a = 0; a < 256; ++a) {
        for (int b = 0; b < 256; ++b) {
            gmul_table[a][b] = gmul(a, b);
        }
    }
}



__device__ void MixColumns(unsigned char* state) {
    unsigned char tmp[16];

    for (int i = 0; i < 4; i++) {
        tmp[i * 4 + 0] = gmul_table[0x02][state[i * 4 + 0]] ^ gmul_table[0x03][state[i * 4 + 1]] ^ state[i * 4 + 2] ^ state[i * 4 + 3];
        tmp[i * 4 + 1] = state[i * 4 + 0] ^ gmul_table[0x02][state[i * 4 + 1]] ^ gmul_table[0x03][state[i * 4 + 2]] ^ state[i * 4 + 3];
        tmp[i * 4 + 2] = state[i * 4 + 0] ^ state[i * 4 + 1] ^ gmul_table[0x02][state[i * 4 + 2]] ^ gmul_table[0x03][state[i * 4 + 3]];
        tmp[i * 4 + 3] = gmul_table[0x03][state[i * 4 + 0]] ^ state[i * 4 + 1] ^ state[i * 4 + 2] ^ gmul_table[0x02][state[i * 4 + 3]];
    }

    for (int i = 0; i < 16; i++) {
        state[i] = tmp[i];
    }
}





__device__ void InvMixColumns(unsigned char* state) {
    unsigned char tmp[16];

    for (int i = 0; i < 4; i++) {
        tmp[i * 4 + 0] = gmul_table[0x0e][state[i * 4 + 0]] ^ gmul_table[0x0b][state[i * 4 + 1]] ^ gmul_table[0x0d][state[i * 4 + 2]] ^ gmul_table[0x09][state[i * 4 + 3]];
        tmp[i * 4 + 1] = gmul_table[0x09][state[i * 4 + 0]] ^ gmul_table[0x0e][state[i * 4 + 1]] ^ gmul_table[0x0b][state[i * 4 + 2]] ^ gmul_table[0x0d][state[i * 4 + 3]];
        tmp[i * 4 + 2] = gmul_table[0x0d][state[i * 4 + 0]] ^ gmul_table[0x09][state[i * 4 + 1]] ^ gmul_table[0x0e][state[i * 4 + 2]] ^ gmul_table[0x0b][state[i * 4 + 3]];
        tmp[i * 4 + 3] = gmul_table[0x0b][state[i * 4 + 0]] ^ gmul_table[0x0d][state[i * 4 + 1]] ^ gmul_table[0x09][state[i * 4 + 2]] ^ gmul_table[0x0e][state[i * 4 + 3]];
    }

    for (int i = 0; i < 16; i++) {
        state[i] = tmp[i];
    }
}





__device__ void AddRoundKey(unsigned char* state, const unsigned char* roundKey) {
    for (int i = 0; i < 16; i++) {
        state[i] ^= roundKey[i];
    }
}




__global__ void initialize_start_key(unsigned char* startKey, unsigned long long seed) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx == 0) { 
        hiprandState state;
        hiprand_init(seed, 0, 0, &state);
        for (int i = 0; i < 16; ++i) {
            startKey[i] = hiprand(&state) % 256;
        }
    }
}









__global__ void generate_sequential_aes_keys(unsigned char* keys, int n, unsigned char* startKey) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx < n) {
        unsigned char carry = 0;
        for (int i = 15; i >= 0; --i) {
            unsigned int keyOffset = (unsigned int)startKey[i] + carry;
            
            if (i == 15) keyOffset += idx;
            keys[idx * 16 + i] = keyOffset % 256;
            carry = keyOffset / 256;
        }

        
        int i = 14;
        while (carry > 0 && i >= 0) {
            unsigned int keyOffset = (unsigned int)keys[idx * 16 + i] + carry;
            keys[idx * 16 + i] = keyOffset % 256;
            carry = keyOffset / 256;
            --i;
        }
    }
}




void increment_key(unsigned char* key, unsigned int n) {
    unsigned int carry = n;
    for (int i = 15; i >= 0 && carry > 0; --i) {
        unsigned int current = key[i] + carry;
        key[i] = current % 256;
        carry = current / 256;
    }
}





__global__ void generate_aes_keys(unsigned char* keys, int n, unsigned long long seed) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx < n) {
        hiprandState state;
        hiprand_init(seed, idx, 0, &state);

        for (int i = 0; i < 16; ++i) {
            keys[idx * 16 + i] = hiprand(&state) % 256;
        }
    }
}



__device__ void AESEncrypt(const unsigned char* plainText, unsigned char* key, unsigned char* cipherText) {
    unsigned char roundKeys[176];
    KeyExpansion(key, roundKeys);

    
    for (int i = 0; i < 16; ++i) {
        cipherText[i] = plainText[i];
    }

    
    AddRoundKey(cipherText, roundKeys);

    
    for (int round = 1; round < 10; ++round) {
        SubBytes(cipherText);
        ShiftRows(cipherText);
        MixColumns(cipherText);
        AddRoundKey(cipherText, roundKeys + round * 16);
    }

    
    SubBytes(cipherText);
    ShiftRows(cipherText);
    AddRoundKey(cipherText, roundKeys + 160);
}




__device__ void AESDecrypt(const unsigned char* cipherText, unsigned char* key, unsigned char* decryptedText) {
    unsigned char roundKeys[176];
    KeyExpansion(key, roundKeys);


    for (int i = 0; i < 16; ++i) {
        decryptedText[i] = cipherText[i];
    }

    AddRoundKey(decryptedText, roundKeys + 160);


    for (int round = 9; round > 0; --round) {
        InvShiftRows(decryptedText);
        InvSubBytes(decryptedText);
        AddRoundKey(decryptedText, roundKeys + round * 16);
        InvMixColumns(decryptedText);
    }


    InvShiftRows(decryptedText);
    InvSubBytes(decryptedText);
    AddRoundKey(decryptedText, roundKeys);
}


__global__ void tryDecryptAES(const unsigned char* cipherText, const unsigned char* expectedPlainText, unsigned char* keys, bool* found, unsigned char* foundKey, unsigned char* decryptedTexts) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    unsigned char decryptedText[16];

    AESDecrypt(cipherText, keys + idx * 16, decryptedText);

    

#if DEBUG_MODE

    for (int i = 0; i < 16; ++i) {
        decryptedTexts[idx * 16 + i] = decryptedText[i];
    }

#endif
   
    
    bool match = true;
    for (int i = 0; i < 16; ++i) {
        if (decryptedText[i] != expectedPlainText[i]) {
            match = false;
            break;
        }
    }
    
    if (match) {
        *found = true;
        for (int i = 0; i < 16; ++i) {
            foundKey[i] = keys[idx * 16 + i];
        }
    }
   
    
}






int main(int argc, char* argv[]) {

    const int n = 116736; // Total de threads    
    int threadsPerBlock = 1024;
    int numberOfBlocks = (n + threadsPerBlock - 1) / threadsPerBlock;

    unsigned long long seed = (unsigned long long)time(NULL);

    unsigned char* d_keys, * d_cipherText, * d_expectedPlainText, * d_foundKey, foundKey[16];
    bool found = false, * d_found;

    unsigned char* d_startKey;

    unsigned char* d_decryptedTexts;
    size_t decryptedTextsSize = n * 16 * sizeof(unsigned char); 
    hipMalloc(&d_decryptedTexts, decryptedTextsSize);




#if DEBUG_MODE
    unsigned char* h_keys = (unsigned char*)malloc(n * 16 * sizeof(unsigned char));
#endif




    //  cipherText and expectedPlainText 
    const unsigned char cipherText[16] = { 0x8B, 0x66, 0x68, 0xC2, 0x7D, 0x22, 0x61, 0x05, 0xA9, 0x17, 0xD6, 0x61, 0x41, 0xBC, 0x7B, 0x67 };
    const unsigned char expectedPlainText[16] = { 0xC4, 0x93, 0xE8, 0x4A, 0xAD, 0xD1, 0xC3, 0x03, 0x91, 0x3A, 0xBD, 0x57, 0xFE, 0x09, 0x79, 0x36 };

    precompute_gmul_table_aes128 << <1, 1 >> > ();
    hipDeviceSynchronize();

    
    hipMalloc(&d_keys, n * 16 * sizeof(unsigned char));
    hipMalloc(&d_found, sizeof(bool));
    hipMalloc(&d_foundKey, 16 * sizeof(unsigned char));
    hipMalloc(&d_cipherText, 16 * sizeof(unsigned char));
    hipMalloc(&d_expectedPlainText, 16 * sizeof(unsigned char));
    hipMalloc(&d_startKey, 16 * sizeof(unsigned char));



    
    hipMemcpy(d_found, &found, sizeof(bool), hipMemcpyHostToDevice);
    hipMemcpy(d_cipherText, cipherText, 16 * sizeof(unsigned char), hipMemcpyHostToDevice);
    hipMemcpy(d_expectedPlainText, expectedPlainText, 16 * sizeof(unsigned char), hipMemcpyHostToDevice);

   

    initialize_start_key << <1, 1 >> > (d_startKey, seed);
    hipDeviceSynchronize();



    unsigned char h_startKey[16];
    hipMemcpy(h_startKey, d_startKey, 16 * sizeof(unsigned char), hipMemcpyDeviceToHost);



    unsigned char startKey[16] = { 0 }; 
    memcpy(startKey, h_startKey, 16); 


    printf("StartKey: ");
    for (int i = 0; i < 16; ++i) {
        printf("%02x ", h_startKey[i]); 
    }
    printf("\n");



#if TIMER
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
#endif

    

    while (!found)
    {

#if TIMER
        hipEventRecord(start);
#endif


        seed++;
        //generate_aes_keys << <numberOfBlocks, threadsPerBlock >> > (d_keys, n, seed);

        hipMemcpy(d_startKey, startKey, 16 * sizeof(unsigned char), hipMemcpyHostToDevice);
        generate_sequential_aes_keys << <numberOfBlocks, threadsPerBlock >> > (d_keys, n, d_startKey);
        hipDeviceSynchronize();
        
        tryDecryptAES << <numberOfBlocks, threadsPerBlock >> > (d_cipherText, d_expectedPlainText, d_keys, d_found, d_foundKey, d_decryptedTexts);
        hipDeviceSynchronize();
        

        hipMemcpy(&found, d_found, sizeof(bool), hipMemcpyDeviceToHost);  


        increment_key(startKey, n);




#if TIMER
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        printf("Execution time for %d threads : %.2f milliseconds\n", n, milliseconds);

#endif






    

#if DEBUG_MODE

        unsigned char* h_decryptedTexts = (unsigned char*)malloc(n * 16 * sizeof(unsigned char));
        hipMemcpy(h_decryptedTexts, d_decryptedTexts, n * 16 * sizeof(unsigned char), hipMemcpyDeviceToHost);
        hipMemcpy(h_keys, d_keys, n * 16 * sizeof(unsigned char), hipMemcpyDeviceToHost);
        hipDeviceSynchronize();
        

        for (int i = 0; i < n; ++i) {
            // Affichage du Cipher constant
            printf("CIPHER:    ");
            for (int j = 0; j < 16; ++j) {
                printf("%02x ", cipherText[j]);  // Supposons que cipherText est disponible ici
            }
            printf("\n");

            // Affichage de la Clé Générée
            printf("KEY:       ");
            for (int j = 0; j < 16; ++j) {
                printf("%02x ", h_keys[i * 16 + j]);
            }
            printf("\n");

            // Affichage du Texte Déchiffré
            printf("DECRYPTED: ");
            for (int j = 0; j < 16; ++j) {
                printf("%02x ", h_decryptedTexts[i * 16 + j]);
            }
            printf("\n\n");
    }

        free(h_decryptedTexts);


#endif



          
    }
    

    if (found) 
	{
        hipMemcpy(foundKey, d_foundKey, 16 * sizeof(unsigned char), hipMemcpyDeviceToHost);
        // Affichage de la clé 
        printf("Key Found !!! : ");
        for (int i = 0; i < 16; ++i) {
            printf("%02x ", foundKey[i]);
        }
        printf("\n");
    }



    

#if DEBUG_MODE
    if (h_keys != NULL) free(h_keys);
#endif 


    hipFree(d_startKey);
    hipFree(d_keys);
    hipFree(d_found);
    hipFree(d_foundKey);
    hipFree(d_cipherText);
    hipFree(d_expectedPlainText);
    hipFree(d_decryptedTexts);

    return 0;
}




